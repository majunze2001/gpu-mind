#include <hip/hip_runtime.h>
#include <iostream>

__global__ void
kernel(int *array, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < N; i += stride)
		array[i] = i;
}

int
main(void)
{
	int N = 1 << 10;
    // 1k ints ~ 2 KB -- 1 page at most
	int *data;
	hipMallocManaged(&data, N * sizeof(int));
	// the devices of compute capability 6.x and greater do not allocate physical memory when calling cudaMallocManaged():
	// in this case physical memory is populated on first touch and may be resident on the CPU or the GPU.

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	// Launch kernel to initialize data, so the data will be on GPU
	kernel<<<blocksPerGrid, threadsPerBlock>>>(data, N);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be i)
	// now on CPU -- show pagefault, causing a page migration from GPU to CPU
	for (int i = 0; i < N; i++)
	{
		if (data[i] != i)
		{
			std::cout << "Error: data[" << i << "] = " << data[i] << "\n";
			return -1;
		}
	}

	std::cout << "Correct!\n";

	// Free memory
	hipFree(data);

	return 0;
}
