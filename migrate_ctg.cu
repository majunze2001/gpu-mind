#include <hip/hip_runtime.h>
#include <iostream>
// #include <limits>
// #include <unistd.h>

__global__ void
kernel(int *array, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < N; i += stride)
		array[i] += i;
}

int
main(void)
{
	// std::cout << "Current PID: " << getpid() << std::endl;
	// std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
	int N = 1 << 10;
    // 1k ints ~ 2 KB -- 1 page at most
	int *data;
	hipMallocManaged(&data, N * sizeof(int));
	// the devices of compute capability 6.x and greater do not allocate physical memory when calling cudaMallocManaged():
	// in this case physical memory is populated on first touch and may be resident on the CPU or the GPU.

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	// Initialize data, so the should be on CPU
	for (int i = 0; i < N; i ++)
		data[i] = i;

	// Launch kernel, this should pagefault, causing a page migration from CPU to GPU
	kernel<<<blocksPerGrid, threadsPerBlock>>>(data, N);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Correctness verification, comment out for shorter trace output
	// a migration from GPU back to CPU
	// Check for errors (all values should be 2i)
	for (int i = 0; i < N; i++)
	{
		if (data[i] != 2 * i)
		{
			std::cout << "Error: data[" << i << "] = " << data[i] << "\n";
			return -1;
		}
	}
	
	std::cout << "Done!\n";

	// Free memory
	hipFree(data);

	return 0;
}
