#include <hip/hip_runtime.h>
#include <iostream>

__global__ void
kernel(int *array, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < N; i += stride)
		array[i] += i;
}

int
main(void)
{
	int N = 1 << 10;
    // 1k ints ~ 2 KB -- 1 page at most
	int *data;
	hipMallocManaged(&data, N * sizeof(int));
	// the devices of compute capability 6.x and greater do not allocate physical memory when calling cudaMallocManaged():
	// in this case physical memory is populated on first touch and may be resident on the CPU or the GPU.

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	// Initialize data, so the data will be on CPU
	for (int i = 0; i < N; i ++)
		data[i] = i;

	// Launch kernel, this should pagefault, causing a page migration from CPU to GPU
	kernel<<<blocksPerGrid, threadsPerBlock>>>(data, N);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// a migration from GPU back to CPU
	// Check for errors (all values should be 2i)
	for (int i = 0; i < N; i++)
	{
		if (data[i] != 2 * i)
		{
			std::cout << "Error: data[" << i << "] = " << data[i] << "\n";
			return -1;
		}
	}

	std::cout << "Correct!\n";

	// Free memory
	hipFree(data);

	return 0;
}
