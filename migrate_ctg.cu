#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel(int *array, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        array[idx] = idx;
    }
}

int main(void)
{
    int N = 1 << 20;
    int *data;
    hipMallocManaged(&data, N*sizeof(int));

    // Launch kernel to initialize data - this will page fault
    kernel<<<1, 256>>>(data, N);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be i)
    for (int i = 0; i < N; i++) {
        if (data[i] != i) {
            std::cout << "Error: data[" << i << "] = " << data[i] << "\n";
            return -1;
        }
    }

    std::cout << "Correct!\n";

    // Free memory
    hipFree(data);

    return 0;
}
